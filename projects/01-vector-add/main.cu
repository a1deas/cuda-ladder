// project 01 - Vector Add - main.cu
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

#define CUDA_OK(stmt) do {                                     \
    hipError_t err = (stmt);                                  \
    if (err != hipSuccess) {                                  \
        fprintf(stderr, "CUDA error %s at %s:%d\n",            \
                hipGetErrorString(err), __FILE__, __LINE__);  \
        return 1;                                              \
    }                                                          \
} while(0)

__global__ void vectorAdd(const int* A, const int* B, int* C, int N) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
    
}

int main() {
    hipEvent_t start, stop;

    CUDA_OK(hipEventCreate(&start));
    CUDA_OK(hipEventCreate(&stop));

    // 20 = estimated 1 million
    // 28 = 268,435,456
    int N = 1 << 28;

    // GPU loves warp multiplicity(32 threads).
    // 256 = 8 warps --> mostly a good choice
    int threads = 256; // optimum
    int blocks = (N + threads - 1) / threads;
    std::vector<int> hA(N), hB(N), hC(N), hRef(N);

    // we should provide device memory
    int *dA=nullptr, *dB=nullptr, *dC=nullptr;
    size_t bytes = N * sizeof(int);
    CUDA_OK(hipMalloc(&dA, bytes));
    CUDA_OK(hipMalloc(&dB, bytes));
    CUDA_OK(hipMalloc(&dC, bytes));
    
    // init arrays
    for (int i = 0; i < N; i++) {
        hA[i] = i;
        hB[i] = 2 * i;
    }

    // copying Host --> Device
    CUDA_OK(hipMemcpy(dA, hA.data(), bytes, hipMemcpyHostToDevice));
    CUDA_OK(hipMemcpy(dB, hB.data(), bytes, hipMemcpyHostToDevice));

    // starting recording
    hipEventRecord(start);
    // launch kernel
    vectorAdd<<<blocks, threads>>>(dA, dB, dC, N);
    // stoping recording
    hipEventRecord(stop);

    // check errors + syncronize
    hipError_t kerr = hipGetLastError();
    if (kerr != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(kerr));
        return 1;
    }
    CUDA_OK(hipEventSynchronize(stop));

    // copying Device --> Host
    CUDA_OK(hipMemcpy(hC.data(), dC, bytes, hipMemcpyDeviceToHost));

    for(int i = 0; i < N; i++) {
        hRef[i] = hA[i] + hB[i];
    }
    bool ok = true;
    for (int i = 0; i < 10; i++) {
        if (hC[i] != hRef[i]) { ok = false; break; }
    }
    printf("Correctness (first 10): %s\n", ok ? "OK" : "MISMATCH");

    // result
    float ms = 0.0f;
    hipEventElapsedTime(&ms, start, stop);
    printf("GPU kernel time: %.3f ms\n", ms);

    // deleting events and freeing some space
    CUDA_OK(hipEventDestroy(start));
    CUDA_OK(hipEventDestroy(stop));
    hipFree(dA); hipFree(dB); hipFree(dC);

    return 0;
}
