// project 02 - Reduction - main.cu
#include <cstdio>
#include <vector>
#include <hip/hip_runtime.h>

// macro to check for the CUDA errors
#define CUDA_OK(stmt) do {                                  \
    hipError_t err = (stmt);                               \
    if (err != hipSuccess) {                               \
        fprintf(stderr, "CUDA error %s at %s:%d\n",         \
            hipGetErrorString(err), __FILE__, __LINE__);   \
        return 1;                                           \
    }                                                       \
} while(0)

// Reduction: Atomic Version
__global__ void reduceAtomic(const int* A, unsigned long long* out, int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;

    unsigned long long local = 0ULL;
    for (int idx = i; idx < N; idx += stride) {
        local += (unsigned long long)A[idx];
    }
    atomicAdd(out, local);
}

// Reduction: Shared Version, one atomic per block
__global__ void reduceShared(const int* A, unsigned long long* out, int N) {
    extern __shared__ unsigned long long result[];
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    int tid = threadIdx.x;

    unsigned long long local = 0ULL;
    for (int idx = i; idx < N; idx += stride) {
        local += (unsigned long long)A[idx];
    }
    result[tid] = local;
    __syncthreads();
    
    // blockDim.x must be %2
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            result[tid] += result[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(out, result[0]);
    }
}

// warp reduction, without divergence
__inline__ __device__
unsigned long long _warpReduceSumULL(unsigned long long v) {
    unsigned mask = 0xFFFFFFFFu;
    v += __shfl_down_sync(mask, v, 16);
    v += __shfl_down_sync(mask, v, 8);
    v += __shfl_down_sync(mask, v, 4);
    v += __shfl_down_sync(mask, v, 2);
    v += __shfl_down_sync(mask, v, 1);
    return v;
}

// Reduction: warp version + shared for step between warps
__global__ void reduceWarp(const int* A, unsigned long long* out, int N) {
    extern __shared__ unsigned long long warpSums[];

    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    int tid = threadIdx.x;
    int lane = tid & 31; // pos in warp
    int wid = tid >> 5;  // warp number in block

    unsigned long long local = 0ULL;
    for (int idx = i; idx < N; idx += stride) {
        local += (unsigned long long)A[idx];
    }

    local = _warpReduceSumULL(local); // reduction on warp
    if (lane == 0) warpSums[wid] = local;
    __syncthreads();

    if (wid == 0) {
        unsigned long long blockSum =
            (lane < (blockDim.x >> 5)) ? warpSums[lane] : 0ULL;
        blockSum = _warpReduceSumULL(blockSum);
        if (lane == 0) atomicAdd(out, blockSum);
    }
}


int main() {    
    hipEvent_t start, stop;
    CUDA_OK(hipEventCreate(&start));
    CUDA_OK(hipEventCreate(&stop));

    int N = 1 << 20; // 1 million will be enough
    int threads = 256;
    int blocks = (N + threads - 1) / threads;
    
    // host data A = 1
    std::vector<int> hA(N, 1);

    // providing memory buffers
    int* dA = nullptr;
    unsigned long long* dOut = nullptr;
    size_t bytes = N * sizeof(int);

    CUDA_OK(hipMalloc(&dA,   bytes));
    CUDA_OK(hipMalloc(&dOut, sizeof(unsigned long long)));

    CUDA_OK(hipMemcpy(dA, hA.data(), bytes, hipMemcpyHostToDevice));


    // Atomic version
    CUDA_OK(hipMemset(dOut, 0, sizeof(unsigned long long)));
    CUDA_OK(hipEventRecord(start));
    reduceAtomic<<<blocks, threads>>>(dA, dOut, N);
    CUDA_OK(hipEventRecord(stop));
    {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "atomic: launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
    }
    CUDA_OK(hipEventSynchronize(stop));
    {
        float ms = 0.0f;
        hipEventElapsedTime(&ms, start, stop);
        unsigned long long hOut = 0ULL;
        CUDA_OK(hipMemcpy(&hOut, dOut, sizeof(unsigned long long), hipMemcpyDeviceToHost));
        printf("atomic: time = %.3f ms | sum = %llu\n", ms, hOut);
    }


    // Shared version
    CUDA_OK(hipMemset(dOut, 0, sizeof(unsigned long long)));
    CUDA_OK(hipEventRecord(start));
    reduceShared<<<blocks, threads, threads * sizeof(unsigned long long)>>>(dA, dOut, N);
    CUDA_OK(hipEventRecord(stop));
    {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "shared: launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
    }
    CUDA_OK(hipEventSynchronize(stop));
    {
        float ms = 0.0f;
        hipEventElapsedTime(&ms, start, stop);
        unsigned long long hOut = 0ULL;
        CUDA_OK(hipMemcpy(&hOut, dOut, sizeof(unsigned long long), hipMemcpyDeviceToHost));
        printf("shared: time = %.3f ms | sum = %llu\n", ms, hOut);
    }


    // Warp version
    CUDA_OK(hipMemset(dOut, 0, sizeof(unsigned long long)));
    CUDA_OK(hipEventRecord(start));
    reduceWarp<<<blocks, threads, (threads/32) * sizeof(unsigned long long)>>>(dA, dOut, N);
    CUDA_OK(hipEventRecord(stop));
    {
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "warp: launch error: %s\n", hipGetErrorString(err));
            return 1;
        }
    }
    CUDA_OK(hipEventSynchronize(stop));
    {
        float ms = 0.0f;
        hipEventElapsedTime(&ms, start, stop);
        unsigned long long hOut = 0ULL;
        CUDA_OK(hipMemcpy(&hOut, dOut, sizeof(unsigned long long), hipMemcpyDeviceToHost));
        printf("warp:   time = %.3f ms | sum = %llu\n", ms, hOut);
    }

    // free and delete events
    CUDA_OK(hipEventDestroy(start));
    CUDA_OK(hipEventDestroy(stop));
    hipFree(dA); // 7. cudaFree
    hipFree(dOut);
    return 0;
}